#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <algorithm>
#include "sift.cuh"

__global__ void gaussianBlurKernel(const unsigned char* img_in, unsigned char* img_out, int width, int height, int channels, const float* kernel, int kernel_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < width && y < height && c < channels) {
        float sum = 0.0f;
        float weight_sum = 0.0f;
        for (int neighbour_y = max(y - (kernel_size / 2), 0); neighbour_y <= min(y + (kernel_size / 2), height - 1); neighbour_y++) {
            for (int neighbour_x = max(x - (kernel_size / 2), 0); neighbour_x <= min(x + (kernel_size / 2), width - 1); neighbour_x++) {
                float w = kernel[(neighbour_y - (y - (kernel_size / 2))) * kernel_size + (neighbour_x - (x - kernel_size / 2))];
                sum += w * img_in[(neighbour_y * width + neighbour_x) * channels + c];
                weight_sum += w;
            }
        }
        img_out[(y * width + x) * channels + c] = (unsigned char)(sum / weight_sum);
    }
}

__global__ void dogKernel(const unsigned char* img1, const unsigned char* img2, float* dog, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < width && y < height && c < channels) {
        int idx = (y * width + x) * channels + c;
        dog[idx] = static_cast<float>(img1[idx]) - static_cast<float>(img2[idx]);
    }
}

Sift::Sift(int num_levels, int kernel_size, const std::vector<float>& sigmas)
    : num_levels_(num_levels), kernel_size_(kernel_size), sigmas_(sigmas) {}

Sift::~Sift() {}


void Sift::createGaussianKernel(float* kernel, int kernel_size, float sigma) {
    int k = kernel_size / 2;
    float sum = 0.0f;
    for (int y = 0; y < kernel_size; ++y) {
        for (int x = 0; x < kernel_size; ++x) {
            float dx = x - k;
            float dy = y - k;
            float value = expf(-(dx * dx + dy * dy) / (2 * sigma * sigma));
            kernel[y * kernel_size + x] = value;
            sum += value;
        }
    }
    for (int i = 0; i < kernel_size * kernel_size; ++i) kernel[i] /= sum;
}

void Sift::gaussianBlur(const unsigned char* img_in, unsigned char* img_out, int width, int height, int channels, const float* kernel, int kernel_size) {
    size_t img_size = width * height * channels * sizeof(unsigned char);
    size_t kernel_bytes = kernel_size * kernel_size * sizeof(float);
    unsigned char *d_in, *d_out;
    float *d_kernel;
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);
    hipMalloc(&d_kernel, kernel_bytes);
    hipMemcpy(d_in, img_in, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_bytes, hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((width+15)/16, (height+15)/16, channels);
    gaussianBlurKernel<<<numBlocks, threadsPerBlock>>>(d_in, d_out, width, height, channels, d_kernel, kernel_size);
    hipDeviceSynchronize();
    hipMemcpy(img_out, d_out, img_size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_kernel);
}

// in: img1, img2
// out: dog
void Sift::differenceOfGaussians(const unsigned char* img1, const unsigned char* img2, float* dog, int width, int height, int channels) {
    size_t img_size = width * height * channels * sizeof(unsigned char);
    size_t dog_size = width * height * channels * sizeof(float);
    unsigned char *d_img1, *d_img2;
    float *d_dog;
    hipMalloc(&d_img1, img_size);
    hipMalloc(&d_img2, img_size);
    hipMalloc(&d_dog, dog_size);
    hipMemcpy(d_img1, img1, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_img2, img2, img_size, hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((width+15)/16, (height+15)/16, channels);
    dogKernel<<<numBlocks, threadsPerBlock>>>(d_img1, d_img2, d_dog, width, height, channels);
    hipDeviceSynchronize();
    hipMemcpy(dog, d_dog, dog_size, hipMemcpyDeviceToHost);
    hipFree(d_img1);
    hipFree(d_img2);
    hipFree(d_dog);
}

//in: img_in
//out: dogs
void Sift::gaussianPyramidAndDoG(const unsigned char* img_in, int width, int height, int channels, int num_levels, const float* sigmas, unsigned char* blurred, float* dogs, int kernel_size) {
    int img_size = width * height * channels;
    std::vector<float> kernel(kernel_size * kernel_size);
    for (int i = 0; i < num_levels; ++i) {
        createGaussianKernel(kernel.data(), kernel_size, sigmas[i]);
        gaussianBlur(img_in, blurred + i * img_size, width, height, channels, kernel.data(), kernel_size);
    }
    for (int i = 0; i < num_levels - 1; ++i) {
        differenceOfGaussians(blurred + i * img_size, blurred + (i + 1) * img_size, dogs + i * img_size, width, height, channels);
    }
}

// std::vector<float> dogs(img_size * (num_levels_ - 1));
void Sift::findDoGKeypoints(const float* dogs, int width, int height, int channels, int num_levels, std::vector<SiftKeypoint>& keypoints, int max_keypoints, float threshold) {
    int img_size = width * height * channels;
    for (int l = 1; l < num_levels - 2; ++l) {
        const float* dog_prev = dogs + (l - 1) * img_size;
        const float* dog_curr = dogs + l * img_size;
        const float* dog_next = dogs + (l + 1) * img_size;
        
        for (int y = 1; y < height - 1; ++y) {
            for (int x = 1; x < width - 1; ++x) {
                for (int c = 0; c < channels; ++c) {
                    int idx = (y * width + x) * channels + c;
                    float val = dog_curr[idx];
                    if (fabs(val) < threshold) continue;
                    bool is_max = true, is_min = true;
                    
                    std::vector<float> neighbours = {
                        dog_prev[((y - 1) * width + (x - 1)) * channels + c],
                        dog_prev[((y - 1) * width + x) * channels + c],
                        dog_prev[((y - 1) * width + (x + 1)) * channels + c],
                        dog_prev[(y * width + (x - 1)) * channels + c],
                        dog_prev[(y * width + x) * channels + c],
                        dog_prev[(y * width + (x + 1)) * channels + c],
                        dog_prev[((y + 1) * width + (x - 1)) * channels + c],
                        dog_prev[((y + 1) * width + x) * channels + c],
                        dog_prev[((y + 1) * width + (x + 1)) * channels + c],

                        dog_next[((y - 1) * width + (x - 1)) * channels + c],
                        dog_next[((y - 1) * width + x) * channels + c],
                        dog_next[((y - 1) * width + (x + 1)) * channels + c],
                        dog_next[(y * width + (x - 1)) * channels + c],
                        dog_next[(y * width + x) * channels + c],
                        dog_next[(y * width + (x + 1)) * channels + c],
                        dog_next[((y + 1) * width + (x - 1)) * channels + c],
                        dog_next[((y + 1) * width + x) * channels + c],
                        dog_next[((y + 1) * width + (x + 1)) * channels + c],

                        dog[((y - 1) * width + (x - 1)) * channels + c],
                        dog[((y - 1) * width + x) * channels + c],
                        dog[((y - 1) * width + (x + 1)) * channels + c],
                        dog[(y * width + (x - 1)) * channels + c],
                        dog[(y * width + (x + 1)) * channels + c],
                        dog[((y + 1) * width + (x - 1)) * channels + c],
                        dog[((y + 1) * width + x) * channels + c],
                        dog[((y + 1) * width + (x + 1)) * channels + c]
                    };

                    for(const auto& nval : neighbours) {
                        if (val <= nval) is_max = false;
                        if (val >= nval) is_min = false;
                    }

                    if ((is_max || is_min) && keypoints.size() < static_cast<size_t>(max_keypoints)) {
                        SiftKeypoint kp;
                        kp.x = x;
                        kp.y = y;
                        kp.scale = l;
                        kp.orientation = 0.0f;
                        kp.descriptor.fill(0.0f);
                        keypoints.push_back(kp);
                    }
                }
            }
        }
    }
}

std::vector<SiftKeypoint> Sift::detectKeypoints(const unsigned char* img_in, int width, int height, int channels) {
    std::vector<SiftKeypoint> keypoints;
    int img_size = width * height * channels;
    std::vector<unsigned char> blurred(img_size * num_levels_);
    std::vector<float> dogs(img_size * (num_levels_ - 1));
    gaussianPyramidAndDoG(img_in, width, height, channels, num_levels_, sigmas_.data(), blurred.data(), dogs.data(), kernel_size_);

    const int max_keypoints = 10000;
    findDoGKeypoints(dogs.data(), width, height, channels, num_levels_, keypoints, max_keypoints, 10.0f);
    return keypoints;
}